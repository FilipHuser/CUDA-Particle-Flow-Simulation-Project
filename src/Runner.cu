#include "hip/hip_runtime.h"
﻿#include <glew.h>
#include <freeglut.h>

// includes, cudaimageWidth
#include <cudaDefs.h>
#include <cuda_gl_interop.h>
#include <hip/hip_vector_types.h>			// normalize method

#include <hiprand/hiprand_kernel.h>


#include <imageManager.h>
#include <imageUtils.cuh>
#include <benchmark.h>

#include "Map.h"
#include "FlowField.h"

#define TPB_1D 32                                  
#define TPB_2D TPB_1D*TPB_1D      
hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();
using DT = uchar4;

int GRID_SIZE = 64;
int NUMBER_OF_PARTICLES = 1024;

constexpr uchar3 backgroundCollor = { 0 , 0 ,0 };
const uchar3 wallCollor = { 0 , 255 , 0 };

const char* vertexShaderSrc = R"(
#version 330 core
layout(location = 0) in vec2 aPosition;
uniform float uPointSize;
void main() {
    gl_Position = vec4(aPosition, 0.0, 1.0);
    gl_PointSize = uPointSize;
}
)";

const char* fragmentShaderSrc = R"(
#version 330 core
uniform vec4 uColor;
out vec4 FragColor;
void main() {
    float dist = length(gl_PointCoord - vec2(0.5));
    if (dist > 0.5) discard;
    FragColor = uColor;
}
)";

GLuint shaderProgram;
GLuint particleVBO;
hipGraphicsResource* cudaVBOResource;


// Struktura pro uchování údajů o OpenGL datech
struct GLData
{
    unsigned int imageWidth;         // Šířka textury
    unsigned int imageHeight;        // Výška textury
    unsigned int imageBPP;           // Počet bitů na pixel (8, 16, 24, nebo 32 bitů)
    unsigned int imagePitch;         // Výška řádku v bitech (počet bajtů na jeden řádek)

    unsigned int pboID;              // ID Pixel Buffer Object (PBO) pro práci s daty textury
    unsigned int textureID;          // ID textury pro OpenGL
    unsigned int viewportWidth = 1024; // Šířka viewportu
    unsigned int viewportHeight = 1024; // Výška viewportu
};

GLData FloatFieldTexture;
GLData RenderTexture;

// Struktura pro uchování údajů pro CUDA texturu a PBO (Pixel Buffer Object)
struct CudaData
{
    hipTextureDesc texDesc;            // Popis textury pro CUDA, obsahuje parametry textury
    hipArray_t texArrayData;           // Data textury ve formátu CUDA
    hipResourceDesc resDesc;           // Popis prostředku pro získání dat z textury
    hipChannelFormatDesc texChannelDesc; // Popis kanálů textury (např. jaké jsou velikosti jednotlivých kanálů)
    hipTextureObject_t texObj;         // CUDA texturový objekt, který bude vytvořen
    hipGraphicsResource_t texResource; // CUDA grafický prostředek pro texturu
    hipGraphicsResource_t pboResource; // CUDA grafický prostředek pro PBO (pro zápis)

    CudaData()
    {
        memset(this, 0, sizeof(CudaData)); // Inicializace všech členů struktury na nulu
    }
};
CudaData FloatFieldCudaData;
CudaData RenderTextureCudaData;


void saveOpenGLTexture(GLuint textureID, int width, int height, const char* filename) {
    glBindTexture(GL_TEXTURE_2D, textureID);
    unsigned char* pixels = new unsigned char[width * height * 4]; // RGBA
    glGetTexImage(GL_TEXTURE_2D, 0, GL_RGBA, GL_UNSIGNED_BYTE, pixels);

    // Flip vertically (OpenGL and FreeImage have opposite Y-axis)
    FIBITMAP* image = FreeImage_ConvertFromRawBits(
        pixels, width, height, width * 4, 32,
        0xFF000000, 0x00FF0000, 0x0000FF00, true
    );

    if (FreeImage_Save(FIF_PNG, image, filename)) {
        std::cout << "Texture saved to: " << filename << std::endl;
    }
    else {
        std::cerr << "Failed to save texture." << std::endl;
    }
    FreeImage_Unload(image);
    delete[] pixels;
}

void display()
{
    // OpenGL Rendering
    glClear(GL_COLOR_BUFFER_BIT);

    // Define the color as a float array (RGBA)
    float color[4] = { 1.0f, 0.0f, 0.0f, 1.0f }; // Red color (RGBA)

    // Get the location of the uniform 'uColor' in the shader
    GLuint colorLoc = glGetUniformLocation(shaderProgram, "uColor");
    GLuint uPointSizeLocation = glGetUniformLocation(shaderProgram, "uPointSize");

    //glUseProgram(shaderProgram);

    // Set the uniform color value using the float array
    glUniform4fv(colorLoc, 1, color);
    glUniform1f(uPointSizeLocation, NUMBER_OF_PARTICLES);

    glEnable(GL_TEXTURE_2D);
    glBindTexture(GL_TEXTURE_2D, RenderTexture.textureID);

    // Draw the quad
    glBegin(GL_QUADS);
    glTexCoord2d(0, 0);          glVertex2d(0, 0);
    glTexCoord2d(1, 0);          glVertex2d(RenderTexture.viewportWidth, 0);
    glTexCoord2d(1, 1);          glVertex2d(RenderTexture.viewportWidth, RenderTexture.viewportHeight);
    glTexCoord2d(0, 1);          glVertex2d(0, RenderTexture.viewportHeight);
    glEnd();

    glDisable(GL_TEXTURE_2D);  // Disable texturing after drawing the quad

    // Swap buffers to display the updated texture
    glutSwapBuffers();
}


void my_resize(GLsizei w, GLsizei h)
{
    FloatFieldTexture.viewportWidth = w;
    FloatFieldTexture.viewportHeight = h;

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    glViewport(0, 0, FloatFieldTexture.viewportWidth, FloatFieldTexture.viewportHeight);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0, FloatFieldTexture.viewportWidth, 0, FloatFieldTexture.viewportHeight);
    
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glutPostRedisplay();
}

__global__ void ConstructMap(hipTextureObject_t texObj, int width, int height, uint8_t* pboData) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;


    if (x < width && y < height) {


        const uchar4 texel = tex2D<uchar4>(texObj, x, y);

        if (texel.z == 1)
        {
            int idx = (y * width + x) * 4;
            pboData[idx] = 0;
            pboData[idx + 1] = 0;
            pboData[idx + 2] = 0; // Red channel (125)
            pboData[idx + 3] = 255; // Alpha channel (255)
        }
        else
        {
            int idx = (y * width + x) * 4;
            pboData[idx] = 255;
            pboData[idx + 1] = 255;
            pboData[idx + 2] = 255; // Red channel (125)
            pboData[idx + 3] = 255; // Alpha channel (255)
        }

        return;
    }
}

void cudaWorker()
{
    // Step 1: Map the CUDA resources (input texture and output PBO)
    hipGraphicsResource_t resources[2] = {
        FloatFieldCudaData.texResource,   // Input texture resource
        RenderTextureCudaData.pboResource // Output PBO resource
    };

    checkCudaErrors(hipGraphicsMapResources(2, resources, 0));

    // Step 2: Get mapped CUDA array and pointer to PBO memory
    uint8_t* pboData = nullptr;
    size_t pboSize = 0;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer(
        (void**)&pboData, &pboSize, RenderTextureCudaData.pboResource));

    if (pboData == nullptr) {
        std::cerr << "Error: PBO pointer is null!" << std::endl;
        return;
    }

    // Step 3: Launch CUDA kernel to modify data in the PBO
    dim3 dimBlock(TPB_1D, TPB_1D, 1);
    dim3 dimGrid((FloatFieldTexture.imageWidth + TPB_1D - 1) / TPB_1D,
        (FloatFieldTexture.imageHeight + TPB_1D - 1) / TPB_1D, 1);

    
 ConstructMap << <dimGrid, dimBlock >> > (
     FloatFieldCudaData.texObj,
     FloatFieldTexture.imageWidth,
     FloatFieldTexture.imageHeight,
     pboData
     );
 

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Step 4: Unmap CUDA resources
    checkCudaErrors(hipGraphicsUnmapResources(2, resources, 0));

    // Step 5: Transfer data from PBO to OpenGL texture
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, RenderTexture.pboID);
    glBindTexture(GL_TEXTURE_2D, RenderTexture.textureID);

    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0,
        RenderTexture.imageWidth, RenderTexture.imageHeight,
        GL_RGBA, GL_UNSIGNED_BYTE, (void*)0);  // Offset into PBO

    GLenum error = glGetError();
    if (error != GL_NO_ERROR) {
        std::cerr << "OpenGL error during texture update: " << error << std::endl;
    }

    // Step 6: Unbind PBO and texture from OpenGL
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
    glBindTexture(GL_TEXTURE_2D, 0);
}


void my_idle()
{
    cudaWorker();

    //saveOpenGLTexture(FloatFieldTexture.textureID, FloatFieldTexture.imageWidth, FloatFieldTexture.imageHeight, "flowField.png");
    //saveOpenGLTexture(RenderTexture.textureID, RenderTexture.imageWidth, RenderTexture.imageHeight, "render.png");

    glutPostRedisplay();
}

GLuint createShader(GLenum type, const char* src) {
    GLuint shader = glCreateShader(type);
    glShaderSource(shader, 1, &src, nullptr);
    glCompileShader(shader);

    // Optional: error checking
    GLint success;
    glGetShaderiv(shader, GL_COMPILE_STATUS, &success);
    if (!success) {
        char infoLog[512];
        glGetShaderInfoLog(shader, 512, nullptr, infoLog);
        std::cerr << "Shader Compile Error:\n" << infoLog << std::endl;
    }

    return shader;
}

GLuint createShaderProgram() {
    GLuint vs = createShader(GL_VERTEX_SHADER, vertexShaderSrc);
    GLuint fs = createShader(GL_FRAGMENT_SHADER, fragmentShaderSrc);

    GLuint program = glCreateProgram();
    glAttachShader(program, vs);
    glAttachShader(program, fs);
    glLinkProgram(program);

    // Optional: program link error checking
    GLint success;
    glGetProgramiv(program, GL_LINK_STATUS, &success);
    if (!success) {
        char infoLog[512];
        glGetProgramInfoLog(program, 512, nullptr, infoLog);
        std::cerr << "Shader Link Error:\n" << infoLog << std::endl;
    }

    glDeleteShader(vs);
    glDeleteShader(fs);
    return program;
}

void initGL(int argc, char** argv)
{
    glutInit(&argc, argv);

    glutInitDisplayMode(GLUT_RGBA | GLUT_DEPTH | GLUT_DOUBLE);
    glutInitWindowSize(FloatFieldTexture.viewportWidth, FloatFieldTexture.viewportHeight);
    glutInitWindowPosition(0, 0);
    glutSetOption(GLUT_RENDERING_CONTEXT, false ? GLUT_USE_CURRENT_CONTEXT : GLUT_CREATE_NEW_CONTEXT);
    glutCreateWindow(0);

    char m_windowsTitle[512];
    sprintf_s(m_windowsTitle, 512, "SimpleView | context %s | renderer %s | vendor %s ",
        (const char*)glGetString(GL_VERSION),
        (const char*)glGetString(GL_RENDERER),
        (const char*)glGetString(GL_VENDOR));
    glutSetWindowTitle(m_windowsTitle);

    glutDisplayFunc(display);
    glutReshapeFunc(my_resize);
    glutIdleFunc(my_idle);
    glutSetCursor(GLUT_CURSOR_CROSSHAIR);

    // initialize necessary OpenGL extensions
    glewInit();

    glClearColor(0.0, 0.0, 0.0, 1.0);
    glShadeModel(GL_SMOOTH);
    glViewport(0, 0, FloatFieldTexture.viewportWidth, FloatFieldTexture.viewportHeight);

    glFlush();
}

void generateFlowFieldTexture(FlowField* flowfield, Map* map)
{
    FloatFieldTexture.imageWidth = flowfield->getSize();
    FloatFieldTexture.imageHeight = flowfield->getSize();

    // Alokování pole pro data textury
    unsigned char* textureData = new unsigned char[FloatFieldTexture.imageWidth * FloatFieldTexture.imageHeight * 4];  // 4 pro RGBA
    for (int y = 0; y < FloatFieldTexture.imageHeight; ++y)
    {
        for (int x = 0; x < FloatFieldTexture.imageWidth; ++x)
        {
            std::array<uint8_t, 2> datapoint = flowfield->getData(x, y);
            int wall = (int)map->getCell(x, y);
            int idx = (y * FloatFieldTexture.imageWidth + x) * 4;  // 4 pro RGBA
            textureData[idx] = datapoint.at(0);
            textureData[idx + 1] = datapoint.at(1);
            textureData[idx + 2] = wall;
            textureData[idx + 3] = 255; // Alpha channel (plně průhledný)
        }
    }

    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &FloatFieldTexture.textureID);
    glBindTexture(GL_TEXTURE_2D, FloatFieldTexture.textureID);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, FloatFieldTexture.imageWidth, FloatFieldTexture.imageHeight, 0, GL_RGBA, GL_UNSIGNED_BYTE, textureData);

    // Nastavení parametrů textury
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);

    // Uvolnění alokované paměti pro texturu
    delete[] textureData;

    // Vytvoření PBO pro OpenGL
    glGenBuffers(1, &FloatFieldTexture.pboID);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, FloatFieldTexture.pboID);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, FloatFieldTexture.imageWidth * FloatFieldTexture.imageHeight * 4, NULL, GL_DYNAMIC_COPY);    // Alokace dat pro PBO
}

void generateRenderTexture() {
    // Initialize the render texture size (same as the flowfield texture in your case)
    RenderTexture.imageWidth = FloatFieldTexture.imageWidth;
    RenderTexture.imageHeight = FloatFieldTexture.imageHeight;

    // Create the OpenGL texture for the render texture
    glGenTextures(1, &RenderTexture.textureID);
    glBindTexture(GL_TEXTURE_2D, RenderTexture.textureID);

    // Create the texture with no initial data, OpenGL will allocate it
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, RenderTexture.imageWidth, RenderTexture.imageHeight, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);

    // Set texture parameters for proper behavior
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);  // Linear filtering for minification
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);  // Linear filtering for magnification
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);  // Clamp texture horizontally
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);  // Clamp texture vertically

    // Create a PBO (Pixel Buffer Object) for OpenGL to use with the texture
    glGenBuffers(1, &RenderTexture.pboID);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, RenderTexture.pboID);

    // Allocate memory for the PBO, where the texture data will be copied into
    glBufferData(GL_PIXEL_UNPACK_BUFFER, RenderTexture.imageWidth * RenderTexture.imageHeight * 4, NULL, GL_DYNAMIC_COPY); // NULL for unused initial data

    // Unbind the PBO to ensure OpenGL operations can proceed safely
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    // After this, the texture and PBO are ready for CUDA integration or further OpenGL operations
}

void initCUDAObjects()
{
    checkCudaErrors(hipGraphicsGLRegisterImage(&FloatFieldCudaData.texResource, FloatFieldTexture.textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly));  // Registrace textury pro CUDA (pro čtení)
    checkCudaErrors(hipGraphicsMapResources(1, &FloatFieldCudaData.texResource, 0));     // Mapování textury pro CUDA
    hipGraphicsSubResourceGetMappedArray(&FloatFieldCudaData.texArrayData, FloatFieldCudaData.texResource, 0, 0);  // Získání datového pole z mapované textury

    // Nastavení popisu pro texturu
    FloatFieldCudaData.resDesc.resType = hipResourceTypeArray;
    FloatFieldCudaData.resDesc.res.array.array = FloatFieldCudaData.texArrayData;
    FloatFieldCudaData.texDesc.readMode = hipReadModeElementType;
    FloatFieldCudaData.texDesc.normalizedCoords = false;
    FloatFieldCudaData.texDesc.filterMode = hipFilterModePoint;
    FloatFieldCudaData.texDesc.addressMode[0] = hipAddressModeClamp;
    FloatFieldCudaData.texDesc.addressMode[1] = hipAddressModeClamp;

    // Získání popisu kanálu textury
    checkCudaErrors(hipGetChannelDesc(&FloatFieldCudaData.texChannelDesc, FloatFieldCudaData.texArrayData));
    // Vytvoření objektu textury v CUDA
    checkCudaErrors(hipCreateTextureObject(&FloatFieldCudaData.texObj, &FloatFieldCudaData.resDesc, &FloatFieldCudaData.texDesc, NULL));
    // Unmapping textury po registraci
    checkCudaErrors(hipGraphicsUnmapResources(1, &FloatFieldCudaData.texResource, 0));
    // Registrace PBO pro zápis do CUDA
    hipGraphicsGLRegisterBuffer(&FloatFieldCudaData.pboResource, FloatFieldTexture.pboID, hipGraphicsRegisterFlagsWriteDiscard);


    //-----------------------


    // Register the RenderTexture texture for CUDA (read-write)
    checkCudaErrors(hipGraphicsGLRegisterImage(&RenderTextureCudaData.texResource, RenderTexture.textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
    checkCudaErrors(hipGraphicsMapResources(1, &RenderTextureCudaData.texResource, 0));  // Map the RenderTexture for CUDA
    hipGraphicsSubResourceGetMappedArray(&RenderTextureCudaData.texArrayData, RenderTextureCudaData.texResource, 0, 0);  // Get the texture data array

    // Set up the texture description for the RenderTexture
    RenderTextureCudaData.resDesc.resType = hipResourceTypeArray;
    RenderTextureCudaData.resDesc.res.array.array = RenderTextureCudaData.texArrayData;
    RenderTextureCudaData.texDesc.readMode = hipReadModeElementType;
    RenderTextureCudaData.texDesc.normalizedCoords = false;
    RenderTextureCudaData.texDesc.filterMode = hipFilterModePoint;
    RenderTextureCudaData.texDesc.addressMode[0] = hipAddressModeClamp;
    RenderTextureCudaData.texDesc.addressMode[1] = hipAddressModeClamp;

    // Get the channel description for the RenderTexture
    checkCudaErrors(hipGetChannelDesc(&RenderTextureCudaData.texChannelDesc, RenderTextureCudaData.texArrayData));

    // Create the texture object in CUDA for RenderTexture
    checkCudaErrors(hipCreateTextureObject(&RenderTextureCudaData.texObj, &RenderTextureCudaData.resDesc, &RenderTextureCudaData.texDesc, NULL));

    // Unmap the RenderTexture after registration
    checkCudaErrors(hipGraphicsUnmapResources(1, &RenderTextureCudaData.texResource, 0));

    // Register the PBO for writing to RenderTexture in CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&RenderTextureCudaData.pboResource, RenderTexture.pboID, hipGraphicsRegisterFlagsWriteDiscard));
}

void createSharedVBO(int numberOfParticles) {
    glGenBuffers(1, &particleVBO);
    glBindBuffer(GL_ARRAY_BUFFER, particleVBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(float2) * numberOfParticles, nullptr, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    hipGraphicsGLRegisterBuffer(&cudaVBOResource, particleVBO, cudaGraphicsMapFlagsWriteDiscard);
}

__global__ void randomizeParticles(float2* particles, int count, float minX, float maxX, float minY, float maxY, unsigned int seed)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= count) return;

    hiprandState state;
    hiprand_init(seed, i, 0, &state);

    float x = minX + (maxX - minX) * hiprand_uniform(&state);
    float y = minY + (maxY - minY) * hiprand_uniform(&state);

    particles[i] = make_float2(x, y);

}
void fillParticlesWithCUDA(int numberOfParticles, float minX, float maxX, float minY, float maxY, unsigned int seed)
{
    float2* dptr;
    hipGraphicsMapResources(1, &cudaVBOResource, 0);
    size_t bytes;
    hipGraphicsResourceGetMappedPointer((void**)&dptr, &bytes, cudaVBOResource);
    int threadsPerBlock = 256;
    int blocks = (numberOfParticles + threadsPerBlock - 1) / threadsPerBlock;
    randomizeParticles << <blocks, threadsPerBlock >> > (dptr, numberOfParticles, minX, maxX, minY, maxY, seed);
    hipGraphicsUnmapResources(1, &cudaVBOResource, 0);
}

void allocateParticles(float2** Dparticles, int numberOfParticles)
{
    hipMalloc((void**)Dparticles, numberOfParticles * sizeof(float2));
}

void releaseOpenGL()
{
    if (FloatFieldTexture.textureID > 0)
        glDeleteTextures(1, &FloatFieldTexture.textureID);
    if (FloatFieldTexture.pboID > 0)
        glDeleteBuffers(1, &FloatFieldTexture.pboID);
}

void releaseCUDA()
{
    checkCudaErrors(hipGraphicsUnregisterResource(FloatFieldCudaData.pboResource));
    checkCudaErrors(hipGraphicsUnregisterResource(FloatFieldCudaData.texResource));
}

void releaseResources()
{
    releaseCUDA();
    releaseOpenGL();
}

int main(int argc, char* argv[])
{
    initializeCUDA(deviceProp);
    FreeImage_Initialise();
    initGL(argc, argv);

    shaderProgram = createShaderProgram();

    Map* map = new Map(32);
    map->setWall(10, 10);
    map->setWall(9, 9);
    map->setWall(8, 8);
    map->setWall(7, 7);
    map->setWall(6, 6);
    map->setWall(5, 5);
    map->setWall(4, 4);
    map->setWall(3, 3);
    map->setStart(2, 2);
    map->setWall(1, 1);
    map->setStart(0, 0);
    map->setGoal(24, 24);

    FlowField* flowfield = new FlowField(32);
    flowfield->generateFlowFieldForMap(map);
    flowfield->printFlowField();

    generateFlowFieldTexture(flowfield, map);
    generateRenderTexture();

    //TEST
    saveOpenGLTexture(FloatFieldTexture.textureID, FloatFieldTexture.imageWidth, FloatFieldTexture.imageHeight, "flowField.png");
    saveOpenGLTexture(RenderTexture.textureID, RenderTexture.imageWidth, RenderTexture.imageHeight, "render.png");

    initCUDAObjects();

    const int PARTICLE_ALPHA = 255;
    const int PARTICLE_WEIGHT = 20;
    const int PARTICLE_RADIUS = 5;

    bool running = true;
    int threadsPerBlock = 256;
    int blocks = (NUMBER_OF_PARTICLES + threadsPerBlock - 1) / threadsPerBlock;
    unsigned int seed = 10;

    float2* Dparticles = nullptr;

    createSharedVBO(NUMBER_OF_PARTICLES);
    fillParticlesWithCUDA(NUMBER_OF_PARTICLES, 0, 10, 0, 10, 10);

    allocateParticles(&Dparticles, NUMBER_OF_PARTICLES);
    randomizeParticles <<<blocks,threadsPerBlock>>> (Dparticles, NUMBER_OF_PARTICLES,-10.0f, 10.0f,-5.0f, 5.0f,seed);
    hipDeviceSynchronize();

    glutMainLoop();

    FreeImage_DeInitialise();
    atexit(releaseResources);

    return 0;
}